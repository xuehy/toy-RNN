#include "hip/hip_runtime.h"
#include "rnn_math.hpp"

template <>
void rnn_gpu_gemv<double>(hipblasHandle_t handle, CBLAS_TRANSPOSE trans,
		  int m, int n,
		  const double *alpha,
		  const double *A, 
		  const double *x,
		  const double *beta,
		  double *y)
{
  hipblasOperation_t cublasTrans = (trans == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasDgemv(handle, cublasTrans, n, m, alpha, A, n, x, 1, beta, y, 1);
}

template <>
void rnn_gpu_gemv<float>(hipblasHandle_t handle, CBLAS_TRANSPOSE trans,
		  int m, int n,
		  const float *alpha,
		  const float *A, 
		  const float *x,
		  const float *beta,
		  float *y)
{
  hipblasOperation_t cublasTrans = (trans == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasSgemv(handle, cublasTrans, n, m, alpha, A, n, x, 1, beta, y, 1);
}

template <>
void rnn_gpu_copy<double>(hipblasHandle_t handle, int N, double *X, double *Y)
{
  hipblasDcopy(handle, N, X, 1, Y, 1);
}

template <>
void rnn_gpu_copy<float>(hipblasHandle_t handle, int N, float *X, float *Y)
{
  hipblasScopy(handle, N, X, 1, Y, 1);
}

template <>
void rnn_gpu_scal<double>(hipblasHandle_t handle, int N, const double *alpha, double *x)
{
  hipblasDscal(handle, N, alpha, x, 1);
}

template <>
void rnn_gpu_scal<float>(hipblasHandle_t handle, int N, const float *alpha, float *x)
{
  hipblasSscal(handle, N, alpha, x, 1);
}

template <>
void rnn_gpu_ger<double>(hipblasHandle_t handle, 
			 int M, int N, double *alpha, double *X, double *Y, double *A)
{
  hipblasDger(handle, N, M, alpha, Y, 1, X, 1, A, N);
}

template <>
void rnn_gpu_ger<float>(hipblasHandle_t handle, 
			 int M, int N, float *alpha, float *X, float *Y, float *A)
{
  hipblasSger(handle, N, M, alpha, Y, 1, X, 1, A, N);
}

template <>
void rnn_gpu_axpy<double>(hipblasHandle_t handle, int N, double *alpha, double *X, double *Y)
{
  hipblasDaxpy(handle, N, alpha, X, 1, Y, 1);
}

template <>
void rnn_gpu_axpy<float>(hipblasHandle_t handle, int N, float *alpha, float *X, float *Y)
{
  hipblasSaxpy(handle, N, alpha, X, 1, Y, 1);
}

template <>
void rnn_gpu_set<int>(hipblasHandle_t handle, int N, const int *X, int *Y)
{
  hipblasSetVector(N, sizeof(int), X, 1, Y, 1);
}

template <>
void rnn_gpu_set<double>(hipblasHandle_t handle, int N, const double *X, double *Y)
{
  hipblasSetVector(N, sizeof(double), X, 1, Y, 1);
}

template <>
void rnn_gpu_set<float>(hipblasHandle_t handle, int N, const float *X, float *Y)
{
  hipblasSetVector(N, sizeof(float), X, 1, Y, 1);
}

template <>
void rnn_gpu_get(hipblasHandle_t handle, int N, const float *X, float *Y)
{
  hipblasGetVector(N, sizeof(float), X, 1, Y, 1);
}

template <>
void rnn_gpu_get(hipblasHandle_t handle, int N, const double *X, double *Y)
{
  hipblasGetVector(N, sizeof(double), X, 1, Y, 1);
}


// cuda kernels
template <typename DTYPE>
__global__ void kernel_exp(DTYPE *input, DTYPE *output, int N)
{
  CUDA_KERNEL_LOOP(index,N)
    {
      output[index] = exp(input[index]);
    }
}

template <typename DTYPE>
__global__ void kernel_add_scalar(DTYPE alpha, DTYPE *input, DTYPE *output, int N)
{
  CUDA_KERNEL_LOOP(index, N)
    {
      output[index] = input[index] + alpha;
    }
}

template <typename DTYPE>
__global__ void kernel_sub_scalar(DTYPE *alpha, const DTYPE *input, DTYPE *output, const int N)
{
  CUDA_KERNEL_LOOP(index, N)
    {
      output[index] = input[index] - alpha[0];
    }
}

template <typename DTYPE>
__global__ void kernel_tanh(const DTYPE *input, DTYPE *output, int N)
{
  CUDA_KERNEL_LOOP(index, N)
    {
      output[index] = tanh(input[index]);
    }
}

template <typename DTYPE>
__global__ void kernel_div_scalar(DTYPE *alpha, DTYPE *input, DTYPE *output, const int N)
{
  CUDA_KERNEL_LOOP(index, N)
    {
      output[index] = input[index] / alpha[0];
    }
}

template <>
void rnn_gpu_tanh<float>(const int N, const float *X, float *Y)
{
  kernel_tanh<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(X, Y, N);
}

template <>
void rnn_gpu_tanh<double>(const int N, const double *X, double *Y)
{
  kernel_tanh<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(X, Y, N);
}

template <typename DTYPE>
__global__ void kernel_max(const DTYPE *input, DTYPE *out, const int N)
{
  CUDA_KERNEL_LOOP(index, 1)
    {
      DTYPE maxval = -FLT_MAX;
      for(int c = 0; c < N; ++c)
	maxval = max(input[c], maxval);
      *out = maxval;
    }
}

template <typename DTYPE>
__global__ void kernel_sum(const DTYPE *input, DTYPE *out, const int N)
{
  CUDA_KERNEL_LOOP(index, 1)
    {
      DTYPE sum = 0;
      for(int c = 0; c < N; ++c)
	sum += input[c];
      *out = sum;
    } 
}

// reduction sum needs detailed improvement
template <typename DTYPE>
__global__ void reduction_sum(const DTYPE *input, DTYPE *output, const int N)
{
      int tid = threadIdx.x;
      int bid = blockIdx.x;
      int index = bid * blockDim.x + tid;
      int block_start = bid * RNN_CUDA_NUM_THREADS;
      // shared memory is shared within the same block
      __shared__ DTYPE s_data[RNN_CUDA_NUM_THREADS];
      if(block_start + tid < N)
	s_data[tid] = input[index];
      else
	s_data[tid] = 0;
	
      __syncthreads();

      
      for(int i = RNN_CUDA_NUM_THREADS / 2; i > 0; i >>= 1)
	{
	  if (tid < i)
	    s_data[tid] = s_data[tid] + s_data[tid + i];
	  __syncthreads();
	}
      if (tid == 0)
	output[bid] = s_data[0];
}

/**
 * takes device pointers as parameters
 * @input 
 * @sum: the array of the same size as input
 */
template <>
void rnn_reduction_sum<float>(const float *input, float *sum, int N)
{
  reduction_sum<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(input, sum, N);
  N = RNN_GET_BLOCKS(N);
  if (N == 1) return;
  while ( N > 1 )
  {
    reduction_sum<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(sum, sum, N);
      N = RNN_GET_BLOCKS(N);
  }
}

template <>
void rnn_reduction_sum<double>(const double *input, double *sum, int N)
{
  reduction_sum<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(input, sum, N);
  N = RNN_GET_BLOCKS(N);
  if( 1 == N ) return;
  while ( N > 1 )
    {
      reduction_sum<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(sum, sum, N);
      N = RNN_GET_BLOCKS(N);
    }
}

// reduction sum needs detailed improvement
template <typename DTYPE>
__global__ void reduction_max(const DTYPE *input, DTYPE *output, const int N)
{
      int tid = threadIdx.x;
      int bid = blockIdx.x;
      int index = bid * blockDim.x + tid;
      int block_start = bid * RNN_CUDA_NUM_THREADS;
      // shared memory is shared within the same block
      __shared__ DTYPE s_data[RNN_CUDA_NUM_THREADS];
      if(block_start + tid < N)
	s_data[tid] = input[index];
      else
	s_data[tid] = -FLT_MAX;
	
      __syncthreads();

      
      for(int i = RNN_CUDA_NUM_THREADS / 2; i > 0; i >>= 1)
	{
	  if (tid < i)
	    s_data[tid] = max(s_data[tid], s_data[tid + i]);
	  __syncthreads();
	}
      if (tid == 0)
	output[bid] = s_data[0];
}

template <>
void rnn_reduction_max<float>(const float *input, float *output, int N)
{
  reduction_max<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(input, output, N);
  N = RNN_GET_BLOCKS(N);
  if (N == 1) return;
  while ( N > 1 )
  {
    reduction_max<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(output, output, N);
      N = RNN_GET_BLOCKS(N);
  }
}

template <>
void rnn_reduction_max<double>(const double *input, double *output, int N)
{
  reduction_max<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(input, output, N);
  N = RNN_GET_BLOCKS(N);
  if (N == 1) return;
  while ( N > 1 )
  {
    reduction_max<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(output, output, N);
      N = RNN_GET_BLOCKS(N);
  }
}

template <>
void rnn_gpu_softmax<float>(const int N, const float *X, float *Y, float *reduction)
{
  rnn_reduction_max<float>(X, reduction, N);
  kernel_sub_scalar<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(reduction, X, Y, N);
  kernel_exp<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(Y, Y, N);
  rnn_reduction_sum<float>(Y, reduction, N);
  kernel_div_scalar<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(reduction, Y, Y, N);

  // float *maxval;
  // hipMalloc((void**)&maxval, sizeof(float));
  // kernel_max<float><<<1,1>>>(X, maxval, N);
  // kernel_sub_scalar<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(maxval, X, Y, N);

  // kernel_exp<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(Y, Y, N);
  // kernel_sum<float><<<1,1>>>(Y, maxval, N);

  // kernel_div_scalar<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(maxval, Y, Y, N);
  // hipFree(maxval)
}

template <>
void rnn_gpu_softmax<double>(const int N, const double *X, double *Y, double *reduction)
{
  rnn_reduction_max<double>(X, reduction, N);
  kernel_sub_scalar<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(reduction, X, Y, N);
  kernel_exp<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(Y, Y, N);
  rnn_reduction_sum<double>(Y, reduction, N);
  kernel_div_scalar<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(reduction, Y, Y, N);
}

// == the following functions are layer-specific
// this function is ugly
template <typename DTYPE>
__global__ void kernel_softmax_grad(DTYPE *input, int *y, const int word_dim, const int T)
{
  CUDA_KERNEL_LOOP(index, T)
    {
      input[index*word_dim + y[index]] -= DTYPE(1.0);
    }
}

template <>
void softmax_grad_gpu<float>(float *input, int *y, const int word_dim, const int T)
{
  kernel_softmax_grad<float><<<RNN_GET_BLOCKS(T), RNN_CUDA_NUM_THREADS>>>(input, y, word_dim, T);
}

template <>
void softmax_grad_gpu<double>(double *input, int *y, const int word_dim, const int T)
{
  kernel_softmax_grad<double><<<RNN_GET_BLOCKS(T), RNN_CUDA_NUM_THREADS>>>(input, y, word_dim, T);
}

template <typename DTYPE>
__global__ void kernel_tanh_grad(DTYPE *input1, DTYPE *input2, DTYPE *output, int N)
{
  CUDA_KERNEL_LOOP(index, N)
    {
      output[index] = input1[index] * (1 - input2[index] * input2[index]);
    }
}

template <>
void tanh_grad_gpu<float>(float *input1, float *input2, float *output, int N)
{
  kernel_tanh_grad<float><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(input1, input2, output ,N);
}

template <>
void tanh_grad_gpu<double>(double *input1, double *input2, double *output, int N)
{
  kernel_tanh_grad<double><<<RNN_GET_BLOCKS(N), RNN_CUDA_NUM_THREADS>>>(input1, input2, output, N);
}

