#include <hipblas.h>
#include <hip/hip_runtime.h>

template <typename DTYPE>
void rnn_gpu_gemv(hipblasHandle_t handle, CblasTranspose trans,
		  int m, int n,
		  const DTYPE *alpha,
		  const DTYPE *A, 
		  const DTYPE *x,
		  const DTYPE *beta,
		  DTYPE *y) {}

template <>
void rnn_gpu_gemv<double>(hipblasHandle_t handle, CblasTranspose trans,
		  int m, int n,
		  const double *alpha,
		  const double *A, 
		  const double *x,
		  const double *beta,
		  double *y)
{
  hipblasOperation_t cublasTrans = (trans == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasDgemv(handle, cublasTrans, n, m, alpha, A, n, x, 1, beta, y, 1);
}

template <>
void rnn_gpu_gemv<float>(hipblasHandle_t handle, CblasTranspose trans,
		  int m, int n,
		  const float *alpha,
		  const float *A, 
		  const float *x,
		  const float *beta,
		  float *y)
{
  hipblasOperation_t cublasTrans = (trans == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasSgemv(handle, cublasTrans, n, m, alpha, A, n, x, 1, beta, y, 1);
}

template <typename DTYPE>
void rnn_gpu_copy(hipblasHandle_t handle, int N, DTYPE *X, DTYPE *Y) {}

template <>
void rnn_gpu_copy<double>(hipblasHandle_t handle, int N, double *X, double *Y)
{
  hipblasDcopy(handle, N, X, 1, Y, 1);
}

template <>
void rnn_gpu_copy<float>(hipblasHandle_t handle, int N, float *X, float *Y)
{
  hipblasScopy(handle, N, X, 1, Y, 1);
}

template <typename DTYPE>
void rnn_gpu_scal<DTYPE>(hipblasHandle_t handle, int N, const DTYPE *alpha, DTYPE *x) {}

template <>
void rnn_gpu_scal<double>(hipblasHandle_t handle, int N, const double *alpha, double *x)
{
  hipblasDscal(handle, N, alpha, x, 1);
}

template <>
void rnn_gpu_scal<float>(hipblasHandle_t handle, int N, const float *alpha, float *x)
{
  hipblasSscal(handle, N, alpha, x, 1);
}
