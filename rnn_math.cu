#include "rnn_math.hpp"

template <>
void rnn_gpu_gemv<double>(hipblasHandle_t handle, CBLAS_TRANSPOSE trans,
		  int m, int n,
		  const double *alpha,
		  const double *A, 
		  const double *x,
		  const double *beta,
		  double *y)
{
  hipblasOperation_t cublasTrans = (trans == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasDgemv(handle, cublasTrans, n, m, alpha, A, n, x, 1, beta, y, 1);
}

template <>
void rnn_gpu_gemv<float>(hipblasHandle_t handle, CBLAS_TRANSPOSE trans,
		  int m, int n,
		  const float *alpha,
		  const float *A, 
		  const float *x,
		  const float *beta,
		  float *y)
{
  hipblasOperation_t cublasTrans = (trans == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasSgemv(handle, cublasTrans, n, m, alpha, A, n, x, 1, beta, y, 1);
}

template <>
void rnn_gpu_copy<double>(hipblasHandle_t handle, int N, double *X, double *Y)
{
  hipblasDcopy(handle, N, X, 1, Y, 1);
}

template <>
void rnn_gpu_copy<float>(hipblasHandle_t handle, int N, float *X, float *Y)
{
  hipblasScopy(handle, N, X, 1, Y, 1);
}

template <>
void rnn_gpu_scal<double>(hipblasHandle_t handle, int N, const double *alpha, double *x)
{
  hipblasDscal(handle, N, alpha, x, 1);
}

template <>
void rnn_gpu_scal<float>(hipblasHandle_t handle, int N, const float *alpha, float *x)
{
  hipblasSscal(handle, N, alpha, x, 1);
}

template <>
void rnn_gpu_ger<double>(hipblasHandle_t handle, 
			 int M, int N, double *alpha, double *X, double *Y, double *A)
{
  hipblasDger(handle, N, M, alpha, Y, 1, X, 1, A, N);
}

template <>
void rnn_gpu_ger<float>(hipblasHandle_t handle, 
			 int M, int N, float *alpha, float *X, float *Y, float *A)
{
  hipblasSger(handle, N, M, alpha, Y, 1, X, 1, A, N);
}

template <>
void rnn_gpu_axpy<double>(hipblasHandle_t handle, int N, double *alpha, double *X, double *Y)
{
  hipblasDaxpy(handle, N, alpha, X, 1, Y, 1);
}

template <>
void rnn_gpu_axpy<float>(hipblasHandle_t handle, int N, float *alpha, float *X, float *Y)
{
  hipblasSaxpy(handle, N, alpha, X, 1, Y, 1);
}